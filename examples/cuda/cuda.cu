#include "hip/hip_runtime.h"
// nvcc cuda_hello.cu -o hello.o
#include <stdio.h>
#define MAX_DEVICE 2
#define RTERROR(status, s)                            \
  if (status != hipSuccess)                          \
  {                                                   \
    printf("%s %s\n", s, hipGetErrorString(status)); \
    hipDeviceReset();                                \
    exit(-1);                                         \
  }

//HelloFromGPU<<<1, 5>>>();
__global__ void HelloFromGPU(void)
{
  printf("Hello from GPU\n");
}

int getDeviceCount() {
  hipError_t status;
  int gpuCount = 0;
  status = hipGetDeviceCount(&gpuCount);
  RTERROR(status, "hipGetDeviceCount failed");
  if (gpuCount == 0)
  {
    printf("No CUDA-capable devices found, exiting.\n");
    hipDeviceReset();
    exit(-1);
  }
  return gpuCount;
}

hipDeviceProp_t getProps(int device)
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  return deviceProp;
}

void cudaGetSetDevice(){
  hipError_t status;
  int device = 0;
  status = hipGetDevice(&device);
  RTERROR(status, "Error fetching current GPU");
  status = hipSetDevice(device);
  RTERROR(status, "Error setting CUDA device");
  hipDeviceSynchronize();
}

void isSupportP2P(int gpuCount)
{
  int uvaOrdinals[MAX_DEVICE];
  int uvaCount = 0;
  int i, j;
  hipDeviceProp_t prop;
  for (i = 0; i < gpuCount; ++i)
  {
    hipGetDeviceProperties(&prop, i);
    if (prop.unifiedAddressing)
    {
      uvaOrdinals[uvaCount] = i;
      printf("   GPU%d \"%15s\"\n", i, prop.name);
      uvaCount += 1;
    }
    else
      printf("   GPU%d \"%15s\"     NOT UVA capable\n", i, prop.name);
  }
  int canAccessPeer_ij, canAccessPeer_ji;
  for (i = 0; i < uvaCount; ++i)
  {
    for (j = i + 1; j < uvaCount; ++j)
    {
      hipDeviceCanAccessPeer(&canAccessPeer_ij, uvaOrdinals[i], uvaOrdinals[j]);
      hipDeviceCanAccessPeer(&canAccessPeer_ji, uvaOrdinals[j], uvaOrdinals[i]);
      if (canAccessPeer_ij * canAccessPeer_ji)
      {
        printf("   GPU%d and GPU%d: YES\n", uvaOrdinals[i], uvaOrdinals[j]);
      }
      else
      {
        printf("   GPU%d and GPU%d: NO\n", uvaOrdinals[i], uvaOrdinals[j]);
      }
    }
  }
}

int main(void)
{
  // get GPU Number
  int gpuCount = getDeviceCount();
  printf("gpucount:%d\n", gpuCount);
  // get SM Number
  hipDeviceProp_t deviceProp = getProps(0);
  printf("SM number:%d\n", deviceProp.multiProcessorCount);
  // get Mode info
  if (deviceProp.computeMode == hipComputeModeDefault)
  {
    printf("GPU is in Compute Mode.\n");
  }
  // get P2P support info
  isSupportP2P(gpuCount);
  return 0;
}
