/*
 * =====================================================================================
 *
 *       Filename:  p2pBandwidthLatencyTest.cu
 *
 *    Description:  This microbenchmark is to obtain the latency & uni/bi-directional
 *                  bandwidth for PCI-e, NVLink-V1 in NVIDIA P100 DGX-1 and NVLink-V2 in 
 *                  V100 DGX-1. Please see our IISWC-18 paper titled "Tartan: Evaluating 
 *                  Modern GPU Interconnects via a Multi-GPU Benchmark Suite". The
 *                  Code is modified from the p2pBandwidthLatencyTest app in 
 *                  NVIDIA CUDA-SDK. Please follow NVIDIA's EULA for end usage. 
 *
 *        Version:  1.0
 *        Created:  01/24/2018 02:12:31 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Ang Li, PNNL
 *        Website:  http://www.angliphd.com  
 *
 * =====================================================================================
 */

/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#define ASCENDING


#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

using namespace std;

const char *sSampleName = "P2P (Peer-to-Peer) GPU Bandwidth Latency Test";

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_SUCCESS);                                           \
        }                                                                 \
    }
__global__ void delay(int * null) {
  float j=threadIdx.x;
  for(int i=1;i<10000;i++)
      j=(j+1)/j;

  if(threadIdx.x == j) null[0] = j;
}

void checkP2Paccess(int numGPUs)
{
    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access,i,j);
                printf("Device=%d %s Access Peer Device=%d\n", i, access ? "CAN" : "CANNOT", j);
            }
        }
    }
    printf("\n***NOTE: In case a device doesn't have P2P access to other one, it falls back to normal memcopy procedure.\nSo you can see lesser Bandwidth (GB/s) in those cases.\n\n");
}

void outputBandwidthMatrix(int numGPUs, bool p2p)
{
    int numElems=10000000;
    int repeat=5;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],numElems*sizeof(int));
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> bandwidthMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            bool routingrequired = false;
            int routingnode = -1;

            if(p2p) {
                hipDeviceCanAccessPeer(&access,i,j);
                if (access)
                {
                    hipDeviceEnablePeerAccess(j,0 );
                    cudaCheckError();
                }
                else if (i != j) //not local communication
                {
                    routingrequired = true;
                    int src2route, route2dst;
#ifdef ASCENDING
                    for (int k=0; k<numGPUs; k++)
#else
                    for (int k=numGPUs-1; k>=0; k--)
#endif
                    {
                        hipDeviceCanAccessPeer(&src2route,i,k);
                        hipDeviceCanAccessPeer(&route2dst,k,j);
                        if (src2route && route2dst)
                        {
                            routingnode =  k;
                            break;
                        }
                    }
                    hipDeviceEnablePeerAccess(routingnode,0 );
                    cudaCheckError();
                    hipSetDevice(routingnode);
                    hipDeviceEnablePeerAccess(j,0 );
                    hipSetDevice(i);
                }
            }

            hipDeviceSynchronize();
            cudaCheckError();

            if (routingrequired)
            {
                delay<<<1,1>>>(NULL);
                hipEventRecord(start[i]);
                for (int r=0; r<repeat; r++)
                {
                    hipMemcpyPeerAsync(buffers[i],i,buffers[routingnode],routingnode,sizeof(int)*numElems);
                    //cudaSetDevice(routingnode);

                    hipMemcpyPeerAsync(buffers[routingnode],routingnode,buffers[j],j,sizeof(int)*numElems);
                    //cudaSetDevice(i);
                }

                hipEventRecord(stop[i]);
                hipDeviceSynchronize();
                cudaCheckError();
            }
            else
            {
                delay<<<1,1>>>(NULL);
                hipEventRecord(start[i]);

                for (int r=0; r<repeat; r++)
                {
                    hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,sizeof(int)*numElems);
                }

                hipEventRecord(stop[i]);
                hipDeviceSynchronize();
                cudaCheckError();
            }

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);
            double time_s=time_ms/1e3;

            double gb=numElems*sizeof(int)*repeat/(double)1e9;
            if(i==j) gb*=2;  //must count both the read and the write here
            bandwidthMatrix[i*numGPUs+j]=gb/time_s;
            if (p2p)
            {
                if (access)
                {
                    hipDeviceDisablePeerAccess(j);
                    cudaCheckError();
                }
                
                if (routingrequired)
                {
                    hipDeviceDisablePeerAccess(routingnode);
                    cudaCheckError();
                    hipSetDevice(routingnode);
                    hipDeviceDisablePeerAccess(j);
                    cudaCheckError();
                    hipSetDevice(i);
                }
            }
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", bandwidthMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
}

void outputBidirectionalBandwidthMatrix(int numGPUs, bool p2p)
{
    int numElems=10000000;
    int repeat=5;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);
    vector<hipStream_t> stream0(numGPUs);
    vector<hipStream_t> stream1(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],numElems*sizeof(int));
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
        hipStreamCreate(&stream0[d]);
        cudaCheckError();
        hipStreamCreate(&stream1[d]);
        cudaCheckError();
    }

    vector<double> bandwidthMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            bool routingrequired = false;
            int routingnode = -1;

            if(p2p) {
                hipDeviceCanAccessPeer(&access,i,j);
                if (access)
                {
                    hipSetDevice(i);
                    hipDeviceEnablePeerAccess(j,0);
                    cudaCheckError();
                    hipSetDevice(j);
                    hipDeviceEnablePeerAccess(i,0);
                    cudaCheckError();
                    hipSetDevice(i);
                }
                else if (i != j) // not the local communication
                {
                    routingrequired = true;
                    int src2route, route2dst;

#ifdef ASCENDING
                    for (int k=0; k<numGPUs; k++)
#else
                    for (int k=numGPUs-1; k>=0; k--)
#endif
                    {
                        hipDeviceCanAccessPeer(&src2route,i,k);
                        hipDeviceCanAccessPeer(&route2dst,k,j);
                        if (src2route && route2dst)
                        {
                            routingnode =  k;
                            break;
                        }
                    }
                    hipSetDevice(i);
                    hipDeviceEnablePeerAccess(routingnode,0 );
                    cudaCheckError();
                    hipSetDevice(routingnode);
                    hipDeviceEnablePeerAccess(i,0 );
                    cudaCheckError();
                    hipDeviceEnablePeerAccess(j,0 );
                    cudaCheckError();
                    hipSetDevice(j);
                    hipDeviceEnablePeerAccess(routingnode,0 );
                    hipSetDevice(i);
                    cudaCheckError();
                }
            }

            hipSetDevice(i);
            hipDeviceSynchronize();
            cudaCheckError();

            if (routingrequired)
            {
                delay<<<1,1>>>(NULL);
                hipEventRecord(start[i]);
                for (int r=0; r<repeat; r++)
                {
                    hipMemcpyPeerAsync(buffers[i],i,buffers[routingnode],routingnode,sizeof(int)*numElems,stream0[i]);
                    hipMemcpyPeerAsync(buffers[j],j,buffers[routingnode],routingnode,sizeof(int)*numElems,stream0[i]);
                    hipMemcpyPeerAsync(buffers[routingnode],routingnode,buffers[j],j,sizeof(int)*numElems,stream0[i]);
                    hipMemcpyPeerAsync(buffers[routingnode],routingnode,buffers[i],i,sizeof(int)*numElems,stream0[i]);
                }

                hipEventRecord(stop[i]);
                hipDeviceSynchronize();
                cudaCheckError();

            }
            else
            {
                delay<<<1,1>>>(NULL);
                hipEventRecord(start[i]);

                for (int r=0; r<repeat; r++)
                {
                    hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,sizeof(int)*numElems,stream0[i]);
                    hipMemcpyPeerAsync(buffers[j],j,buffers[i],i,sizeof(int)*numElems,stream1[i]);
                }

                hipEventRecord(stop[i]);
                hipDeviceSynchronize();
                cudaCheckError();
            }




            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);
            double time_s=time_ms/1e3;

            double gb=2.0*numElems*sizeof(int)*repeat/(double)1e9;
            if(i==j) gb*=2;  //must count both the read and the write here
            bandwidthMatrix[i*numGPUs+j]=gb/time_s;
            if(p2p)
            {
                if (access)
                {
                    hipSetDevice(i);
                    hipDeviceDisablePeerAccess(j);
                    hipSetDevice(j);
                    hipDeviceDisablePeerAccess(i);
                }
                
                if (routingrequired)
                {
                    hipSetDevice(i);
                    hipDeviceDisablePeerAccess(routingnode);
                    hipSetDevice(routingnode);
                    hipDeviceDisablePeerAccess(i);
                    hipDeviceDisablePeerAccess(j);
                    hipSetDevice(j);
                    hipDeviceDisablePeerAccess(routingnode);
                    hipSetDevice(i);
                }
            }
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", bandwidthMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
        hipStreamDestroy(stream0[d]);
        cudaCheckError();
        hipStreamDestroy(stream1[d]);
        cudaCheckError();
    }
}

void outputLatencyMatrix(int numGPUs, bool p2p)
{
    int repeat=10000;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],1);
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> latencyMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);
        for (int j=0; j<numGPUs; j++)
        {
            int access;
            bool routingrequired = false;
            int routingnode = -1;
            if(p2p) 
            {
                hipDeviceCanAccessPeer(&access,i,j);
                if (access)
                {
                    hipDeviceEnablePeerAccess(j,0);
                    cudaCheckError();
                }
                else if (i!=j) //not local communication
                {
                    routingrequired = true;
                    int src2route, route2dst;
#ifdef ASCENDING
                    for (int k=0; k<numGPUs; k++)
#else
                    for (int k=numGPUs-1; k>=0; k--)
#endif
                    {
                        hipDeviceCanAccessPeer(&src2route,i,k);
                        hipDeviceCanAccessPeer(&route2dst,k,j);
                        if (src2route && route2dst)
                        {
                            routingnode =  k;
                            break;
                        }
                    }
                    hipSetDevice(i);
                    hipDeviceEnablePeerAccess(routingnode,0 );
                    cudaCheckError();
                    hipSetDevice(routingnode);
                    hipDeviceEnablePeerAccess(j,0 );
                    cudaCheckError();
                    hipSetDevice(i);
                }
            }
            hipDeviceSynchronize();
            cudaCheckError();


            if (routingrequired)
            {
                delay<<<1,1>>>(NULL);
                hipEventRecord(start[i]);

                for (int r=0; r<repeat; r++)
                {
                    hipMemcpyPeerAsync(buffers[i],i,buffers[routingnode],routingnode,1);
                    hipMemcpyPeerAsync(buffers[routingnode],routingnode,buffers[j],j,1);
                }

                hipEventRecord(stop[i]);
                hipDeviceSynchronize();
                cudaCheckError();
            }
            else
            {
                delay<<<1,1>>>(NULL);
                hipEventRecord(start[i]);

                for (int r=0; r<repeat; r++)
                {
                    hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,1);
                }

                hipEventRecord(stop[i]);
                hipDeviceSynchronize();
                cudaCheckError();
            
            }

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);

            latencyMatrix[i*numGPUs+j]=time_ms*1e3/repeat;
            if(p2p)
            {
                if (access)
                {
                    hipDeviceDisablePeerAccess(j);
                    cudaCheckError();
                }
                if (routingrequired)
                {
                    //printf("%d=>%d=>%d,(access:%d,routingrequired:%d\n",i,routingnode,j,access, routingrequired);
                    cudaCheckError();
                    hipDeviceDisablePeerAccess(routingnode);
                    cudaCheckError();
                    hipSetDevice(routingnode);
                    hipDeviceDisablePeerAccess(j);
                    cudaCheckError();
                    hipSetDevice(i);
                }
            }
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", latencyMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
}

int main(int argc, char **argv)
{

    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    printf("[%s]\n", sSampleName);

    //output devices
    for (int i=0; i<numGPUs; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);
        printf("Device: %d, %s, pciBusID: %x, pciDeviceID: %x, pciDomainID:%x\n",i,prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);
    }

    checkP2Paccess(numGPUs);

    //Check peer-to-peer connectivity
    printf("P2P Connectivity Matrix\n");
    printf("     D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d", j);
    }
    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d\t", i);
        for (int j=0; j<numGPUs; j++)
        {
            if (i!=j)
            {
               int access;
               hipDeviceCanAccessPeer(&access,i,j);
               printf("%6d", (access) ? 1 : 0);
            }
            else
            {
                printf("%6d", 1);
            }
        }
        printf("\n");
    }

    printf("Unidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
    outputBandwidthMatrix(numGPUs, false);
    printf("Unidirectional P2P=Enabled Bandwidth Matrix (GB/s)\n");
    outputBandwidthMatrix(numGPUs, true);
    printf("Bidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
    outputBidirectionalBandwidthMatrix(numGPUs, false);
    printf("Bidirectional P2P=Enabled Bandwidth Matrix (GB/s)\n");
    outputBidirectionalBandwidthMatrix(numGPUs, true);


    printf("P2P=Disabled Latency Matrix (us)\n");
    outputLatencyMatrix(numGPUs, false);
    printf("P2P=Enabled Latency Matrix (us)\n");
    outputLatencyMatrix(numGPUs, true);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    exit(EXIT_SUCCESS);
}
